#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "util_cuda.hpp"

#define PI 3.14159265358979323846

CudaUtil::CudaUtil(int edge_set_num_points, int normal_accuracy_num_points) {
    // Allocate memory for edge set
    _edge_set_data.point_cloud_size = edge_set_num_points;
    hipMalloc(&_edge_set_data.point_cloud, edge_set_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_edge_set_data.normals, edge_set_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_edge_set_data.edge_mask_cuda, edge_set_num_points*sizeof(char));
    _edge_set_data.edge_mask_host = new char[edge_set_num_points];

    // Allocate memory for normal accuracy
    _angles_data.normals_size = normal_accuracy_num_points;
    hipMalloc(&_angles_data.out_points, normal_accuracy_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_angles_data.out_normals, normal_accuracy_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_angles_data.gt_points, normal_accuracy_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_angles_data.gt_normals, normal_accuracy_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_angles_data.angles_cuda, normal_accuracy_num_points*sizeof(float));
    _angles_data.angles_host = new float[normal_accuracy_num_points];
}

CudaUtil::~CudaUtil() {
    // Free memory for edge set
    hipFree(_edge_set_data.point_cloud);
    hipFree(_edge_set_data.normals);
    hipFree(_edge_set_data.edge_mask_cuda);
    delete[] _edge_set_data.edge_mask_host;

    // Free memory for normal accuracy
    hipFree(_angles_data.out_normals);
    hipFree(_angles_data.gt_normals);
    hipFree(_angles_data.angles_cuda);
    delete[] _angles_data.angles_host;
}

__global__ void cu_compute_edge_set(Eigen::Vector3f* point_cloud, Eigen::Vector3f* normals, char* out_mask, size_t num_points, float normal_epsilon, float distance_epsilon) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_points) return;

    Eigen::Vector3f p = point_cloud[idx];
    Eigen::Vector3f n = normals[idx];

    char is_edge = 0;
    for (int i = 0; i < num_points; i++) {
        if (i == idx) continue;
        Eigen::Vector3f q = point_cloud[i];
        Eigen::Vector3f r = normals[i];

        float dist = (q-p).norm();
        if (dist > distance_epsilon) continue;

        float dot = n.dot(r);
        if (abs(dot) > normal_epsilon) continue;

        is_edge = 1;
        break;
    }

    out_mask[idx] = is_edge;
}

std::vector<int> CudaUtil::compute_edge_set(const Points& point_cloud, const Points& normals, float normal_epsilon, float distance_epsilon) {

    size_t num_points = point_cloud.rows();

    // CUDA pointers
    Eigen::Vector3f* point_cloud_device;
    Eigen::Vector3f* normals_device;
    char* out_mask_device;
    
    // Host pointers
    char* out_mask;

    if(num_points != _edge_set_data.point_cloud_size) {
        hipMalloc(&point_cloud_device, num_points*sizeof(Eigen::Vector3f));
        hipMalloc(&normals_device, num_points*sizeof(Eigen::Vector3f));
        hipMalloc(&out_mask_device, num_points*sizeof(char));
        out_mask = new char[num_points];
    } else {
        point_cloud_device = _edge_set_data.point_cloud;
        normals_device = _edge_set_data.normals;
        out_mask_device = _edge_set_data.edge_mask_cuda;
        out_mask = _edge_set_data.edge_mask_host;
    }

    hipMemcpy(point_cloud_device, point_cloud.data(), num_points*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(normals_device, normals.data(), num_points*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);

    // Run kernel
    cu_compute_edge_set<<<(num_points+1023)/1024,1024>>>(point_cloud_device, normals_device, out_mask_device, num_points, normal_epsilon, distance_epsilon);

    // Copy back
    hipMemcpy(out_mask, out_mask_device, num_points*sizeof(char), hipMemcpyDeviceToHost);

    std::vector<int> ret;
    for(size_t i = 0; i < num_points; i++) {
        if (out_mask[i] == 1) ret.push_back(i); 
    }

    // Free memory
    if(num_points != _edge_set_data.point_cloud_size) {
        hipFree(point_cloud_device);
        hipFree(normals_device);
        hipFree(out_mask_device);
        delete[] out_mask;
    }

    return ret;
}

__global__ void cu_compute_angles(Eigen::Vector3f* out_points, Eigen::Vector3f* out_normals, Eigen::Vector3f* gt_points, Eigen::Vector3f* gt_normals, float* angles, size_t num_normals) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_normals) return;

    Eigen::Vector3f n = out_normals[idx];
    Eigen::Vector3f gt_n;
    
    // Find closest point
    float min_dist = -1.0;
    int min_idx = -1;
    for (int i = 0; i < num_normals; i++) {
        float dist = (out_points[idx] - gt_points[i]).norm();
        if (dist < min_dist || min_dist < 0.0) {
            min_dist = dist;
            min_idx = i;
        }
    }

    gt_n = gt_normals[min_idx];

    // Calculate angle
    float dot = n.dot(gt_n);
    float norm_dot = dot / (n.norm() * gt_n.norm());
    angles[idx] = acos(fmax(fmin(norm_dot, 1.0f), -1.0f)) * 180.0 / PI;
}

Eigen::VectorXf CudaUtil::compute_angles(const Points& out_points, const Points& out_normals, const Points& gt_points, const Points& gt_normals, float angle_threshold) {
    size_t num_normals = out_normals.rows();

    // CUDA pointers
    Eigen::Vector3f* out_points_device;
    Eigen::Vector3f* out_normals_device;
    Eigen::Vector3f* gt_points_device;
    Eigen::Vector3f* gt_normals_device;
    float* out_angle_device;

    // Host pointers
    float* out_angle;

    if(num_normals != _angles_data.normals_size) {
        hipMalloc(&out_points_device, num_normals*sizeof(Eigen::Vector3f));
        hipMalloc(&out_normals_device, num_normals*sizeof(Eigen::Vector3f));
        hipMalloc(&gt_points_device, num_normals*sizeof(Eigen::Vector3f));
        hipMalloc(&gt_normals_device, num_normals*sizeof(Eigen::Vector3f));
        hipMalloc(&out_angle_device, num_normals*sizeof(float));
        out_angle = new float[num_normals];
    } else {
        out_points_device = _angles_data.out_points;
        out_normals_device = _angles_data.out_normals;
        gt_points_device = _angles_data.gt_points; 
        gt_normals_device = _angles_data.gt_normals;
        out_angle_device = _angles_data.angles_cuda;
        out_angle = _angles_data.angles_host;
    }

    hipMemcpy(out_points_device, out_points.data(), num_normals*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(out_normals_device, out_normals.data(), num_normals*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(gt_points_device, gt_points.data(), num_normals*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(gt_normals_device, gt_normals.data(), num_normals*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);

    // Run kernel
    cu_compute_angles<<<(num_normals+1023)/1024,1024>>>(out_points_device, out_normals_device, gt_points_device, gt_normals_device, out_angle_device, num_normals);

    // Copy back
    hipMemcpy(out_angle, out_angle_device, num_normals*sizeof(float), hipMemcpyDeviceToHost);

    Eigen::VectorXf ret(num_normals);
    for(size_t i = 0; i < num_normals; i++) {
        ret(i) = abs(out_angle[i]);
    }

    // Free memory
    if(num_normals != _angles_data.normals_size) {
        hipFree(out_points_device);
        hipFree(out_normals_device);
        hipFree(gt_points_device);
        hipFree(gt_normals_device);
        hipFree(out_angle_device);
        delete[] out_angle;
    }

    return ret;
}

__global__ void cu_compute_normal_accuracy(Eigen::Vector3f* vertices, Eigen::Vector3i* indices, float* aspect_ratios, float* min_angles, size_t num_triangles) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_triangles) return;

    Eigen::Vector3f v0 = vertices[indices[idx](0)];
    Eigen::Vector3f v1 = vertices[indices[idx](1)];
    Eigen::Vector3f v2 = vertices[indices[idx](2)];

    float a = (v1 - v0).norm();
    float b = (v2 - v0).norm();
    float c = (v2 - v1).norm();

    // Aspect ratio
    float s = (a + b + c) / 2.0f;
    aspect_ratios[idx] = (a * b * c) / (8.0f * (s-a) * (s-b) * (s-c));

    // Min angle
    float min_angle = 0.0f;
    if(a <= b && a <= c) {
        float cos_a = fmax(fmin((b*b + c*c - a*a) / (2.0f * b * c), 1.0f), -1.0f);
        min_angle = acos(cos_a) * 180.0 / PI;
    } else if(b <= a && b <= c) {
        float cos_b = fmax(fmin((a*a + c*c - b*b) / (2.0f * a * c), 1.0f), -1.0f);
        min_angle = acos(cos_b) * 180.0 / PI;
    } else {
        float cos_c = fmax(fmin((b*b + a*a - c*c) / (2.0f * b * a), 1.0f), -1.0f);
        min_angle = acos(cos_c) * 180.0 / PI;
    }
    min_angles[idx] = min_angle;
}

std::pair<Eigen::VectorXf, Eigen::VectorXf> CudaUtil::compute_aspect_ratios_and_min_angles(const TriangleMesh& mesh) {
    size_t num_triangles = mesh.indices.rows();

    // CUDA pointers
    Eigen::Vector3f* vertices_device;
    Eigen::Vector3i* indices_device;
    float* out_aspect_ratios_device;
    float* out_min_angles_device;

    // Host pointers
    float* out_aspect_ratios;
    float* out_min_angles;

    hipMalloc(&vertices_device, mesh.vertices.rows()*sizeof(Eigen::Vector3f));
    hipMalloc(&indices_device, num_triangles*sizeof(Eigen::Vector3i));
    hipMalloc(&out_aspect_ratios_device, num_triangles*sizeof(float));
    hipMalloc(&out_min_angles_device, num_triangles*sizeof(float));
    out_aspect_ratios = new float[num_triangles];
    out_min_angles = new float[num_triangles];

    hipMemcpy(vertices_device, mesh.vertices.data(), mesh.vertices.rows()*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(indices_device, mesh.indices.data(), num_triangles*sizeof(Eigen::Vector3i), hipMemcpyHostToDevice);

    // Run kernel
    cu_compute_normal_accuracy<<<(num_triangles+1023)/1024,1024>>>(vertices_device, indices_device, out_aspect_ratios_device, out_min_angles_device, num_triangles);

    // Copy back
    hipMemcpy(out_aspect_ratios, out_aspect_ratios_device, num_triangles*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(out_min_angles, out_min_angles_device, num_triangles*sizeof(float), hipMemcpyDeviceToHost);

    Eigen::VectorXf aspect_ratios(num_triangles);
    Eigen::VectorXf min_angles(num_triangles);
    for(size_t i = 0; i < num_triangles; i++) {
        aspect_ratios(i) = out_aspect_ratios[i];
        min_angles(i) = out_min_angles[i];
    }

    // Free memory
    hipFree(vertices_device);
    hipFree(indices_device);
    hipFree(out_aspect_ratios_device);
    hipFree(out_min_angles_device);
    delete[] out_aspect_ratios;
    delete[] out_min_angles;

    return std::make_pair(aspect_ratios, min_angles);
}
