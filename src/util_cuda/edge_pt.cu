#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "util_cuda.hpp"

#define PI 3.14159265358979323846

CudaUtil::CudaUtil(int edge_set_num_points, int normal_accuracy_num_points) {
    // Allocate memory for edge set
    _edge_set_data.point_cloud_size = edge_set_num_points;
    hipMalloc(&_edge_set_data.point_cloud, edge_set_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_edge_set_data.normals, edge_set_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_edge_set_data.edge_mask_cuda, edge_set_num_points*sizeof(char));
    _edge_set_data.edge_mask_host = new char[edge_set_num_points];

    // Allocate memory for normal accuracy
    _angles_data.normals_size = normal_accuracy_num_points;
    hipMalloc(&_angles_data.out_points, normal_accuracy_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_angles_data.out_normals, normal_accuracy_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_angles_data.gt_points, normal_accuracy_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_angles_data.gt_normals, normal_accuracy_num_points*sizeof(Eigen::Vector3f));
    hipMalloc(&_angles_data.angles_cuda, normal_accuracy_num_points*sizeof(float));
    _angles_data.angles_host = new float[normal_accuracy_num_points];
}

CudaUtil::~CudaUtil() {
    // Free memory for edge set
    hipFree(_edge_set_data.point_cloud);
    hipFree(_edge_set_data.normals);
    hipFree(_edge_set_data.edge_mask_cuda);
    delete[] _edge_set_data.edge_mask_host;

    // Free memory for normal accuracy
    hipFree(_angles_data.out_normals);
    hipFree(_angles_data.gt_normals);
    hipFree(_angles_data.angles_cuda);
    delete[] _angles_data.angles_host;
}

__global__ void cu_compute_edge_set(Eigen::Vector3f* point_cloud, Eigen::Vector3f* normals, char* out_mask, size_t num_points, float normal_epsilon, float distance_epsilon) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_points) return;

    Eigen::Vector3f p = point_cloud[idx];
    Eigen::Vector3f n = normals[idx];

    char is_edge = 0;
    for (int i = 0; i < num_points; i++) {
        if (i == idx) continue;
        Eigen::Vector3f q = point_cloud[i];
        Eigen::Vector3f r = normals[i];

        float dist = (q-p).norm();
        if (dist > distance_epsilon) continue;

        float dot = n.dot(r);
        if (abs(dot) > normal_epsilon) continue;

        is_edge = 1;
        break;
    }

    out_mask[idx] = is_edge;
}

std::vector<int> CudaUtil::compute_edge_set(const Points& point_cloud, const Points& normals, float normal_epsilon, float distance_epsilon) {

    size_t num_points = point_cloud.rows();

    // CUDA pointers
    Eigen::Vector3f* point_cloud_device;
    Eigen::Vector3f* normals_device;
    char* out_mask_device;
    
    // Host pointers
    char* out_mask;

    if(num_points != _edge_set_data.point_cloud_size) {
        hipMalloc(&point_cloud_device, num_points*sizeof(Eigen::Vector3f));
        hipMalloc(&normals_device, num_points*sizeof(Eigen::Vector3f));
        hipMalloc(&out_mask_device, num_points*sizeof(char));
        out_mask = new char[num_points];
    } else {
        point_cloud_device = _edge_set_data.point_cloud;
        normals_device = _edge_set_data.normals;
        out_mask_device = _edge_set_data.edge_mask_cuda;
        out_mask = _edge_set_data.edge_mask_host;
    }

    hipMemcpy(point_cloud_device, point_cloud.data(), num_points*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(normals_device, normals.data(), num_points*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);

    // Run kernel
    cu_compute_edge_set<<<(num_points+1023)/1024,1024>>>(point_cloud_device, normals_device, out_mask_device, num_points, normal_epsilon, distance_epsilon);

    // Copy back
    hipMemcpy(out_mask, out_mask_device, num_points*sizeof(char), hipMemcpyDeviceToHost);

    std::vector<int> ret;
    for(size_t i = 0; i < num_points; i++) {
        if (out_mask[i] == 1) ret.push_back(i); 
    }

    // Free memory
    if(num_points != _edge_set_data.point_cloud_size) {
        hipFree(point_cloud_device);
        hipFree(normals_device);
        hipFree(out_mask_device);
        delete[] out_mask;
    }

    return ret;
}

__global__ void cu_compute_normal_accuracy(Eigen::Vector3f* out_points, Eigen::Vector3f* out_normals, Eigen::Vector3f* gt_points, Eigen::Vector3f* gt_normals, float* angles, size_t num_normals) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_normals) return;

    Eigen::Vector3f n = out_normals[idx];
    Eigen::Vector3f gt_n;
    
    // Find closest point
    float min_dist = -1.0;
    int min_idx = -1;
    for (int i = 0; i < num_normals; i++) {
        float dist = (out_points[idx] - gt_points[i]).norm();
        if (dist < min_dist || min_dist < 0.0) {
            min_dist = dist;
            min_idx = i;
        }
    }

    gt_n = gt_normals[min_idx];

    // Calculate angle
    float dot = n.dot(gt_n);
    float norm_dot = dot / (n.norm() * gt_n.norm());
    angles[idx] = acos(norm_dot) * 180.0 / PI;
}

Eigen::VectorXf CudaUtil::compute_angles(const Points& out_points, const Points& out_normals, const Points& gt_points, const Points& gt_normals, float angle_threshold) {
    size_t num_normals = out_normals.rows();

    // CUDA pointers
    Eigen::Vector3f* out_points_device;
    Eigen::Vector3f* out_normals_device;
    Eigen::Vector3f* gt_points_device;
    Eigen::Vector3f* gt_normals_device;
    float* out_angle_device;

    // Host pointers
    float* out_angle;

    if(num_normals != _angles_data.normals_size) {
        hipMalloc(&out_points_device, num_normals*sizeof(Eigen::Vector3f));
        hipMalloc(&out_normals_device, num_normals*sizeof(Eigen::Vector3f));
        hipMalloc(&gt_points_device, num_normals*sizeof(Eigen::Vector3f));
        hipMalloc(&gt_normals_device, num_normals*sizeof(Eigen::Vector3f));
        hipMalloc(&out_angle_device, num_normals*sizeof(float));
        out_angle = new float[num_normals];
    } else {
        out_points_device = _angles_data.out_points;
        out_normals_device = _angles_data.out_normals;
        gt_points_device = _angles_data.gt_points; 
        gt_normals_device = _angles_data.gt_normals;
        out_angle_device = _angles_data.angles_cuda;
        out_angle = _angles_data.angles_host;
    }

    hipMemcpy(out_points_device, out_points.data(), num_normals*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(out_normals_device, out_normals.data(), num_normals*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(gt_points_device, gt_points.data(), num_normals*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);
    hipMemcpy(gt_normals_device, gt_normals.data(), num_normals*sizeof(Eigen::Vector3f), hipMemcpyHostToDevice);

    // Run kernel
    cu_compute_normal_accuracy<<<(num_normals+1023)/1024,1024>>>(out_points_device, out_normals_device, gt_points_device, gt_normals_device, out_angle_device, num_normals);

    // Copy back
    hipMemcpy(out_angle, out_angle_device, num_normals*sizeof(float), hipMemcpyDeviceToHost);

    Eigen::VectorXf ret(num_normals);
    for(size_t i = 0; i < num_normals; i++) {
        ret(i) = abs(out_angle[i]);
    }
    return ret;
}
